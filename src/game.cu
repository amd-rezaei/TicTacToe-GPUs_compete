#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include <time.h>

#define DEFAULT_ROWS 3
#define DEFAULT_COLUMNS 3

// Kernel for Player 1: Makes a random move
__global__ void randomMove(int *board, int player, int rows, int columns, unsigned long long seed)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx == 0)
    {
        hiprandState_t state;
        hiprand_init(seed, idx, 0, &state);

        int col = hiprand(&state) % columns;
        int startCol = col;

        while (true)
        {
            if (board[col * rows] == 0)
            {
                for (int i = rows - 1; i >= 0; i--)
                {
                    if (board[col * rows + i] == 0)
                    {
                        board[col * rows + i] = player;
                        return;
                    }
                }
            }
            col = (col + 1) % columns;
            if (col == startCol)
                break; // All columns checked
        }
    }
}

__global__ void lookaheadMove(int *board, int player, int rows, int columns)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx == 0)
    {
        int opponent = 3 - player;

        // Check for a potential win on the main diagonal (top-left to bottom-right)
        int main_diag_count = 0;
        int empty_main_diag_idx = -1;
        for (int diag = 0; diag < rows; diag++)
        {
            int index = diag * columns + diag;
            if (board[index] == opponent)
            {
                main_diag_count++;
            }
            else if (board[index] == 0)
            {
                empty_main_diag_idx = index;
            }
        }
        if (main_diag_count == rows - 1 && empty_main_diag_idx != -1)
        {
            board[empty_main_diag_idx] = player;
            return;
        }

        // Check for a potential win on the anti-diagonal (top-right to bottom-left)
        int anti_diag_count = 0;
        int empty_anti_diag_idx = -1;
        for (int diag = 0; diag < rows; diag++)
        {
            int index = diag * columns + (columns - 1 - diag);
            if (board[index] == opponent)
            {
                anti_diag_count++;
            }
            else if (board[index] == 0)
            {
                empty_anti_diag_idx = index;
            }
        }
        if (anti_diag_count == rows - 1 && empty_anti_diag_idx != -1)
        {
            board[empty_anti_diag_idx] = player;
            return;
        }

        // Check for a potential horizontal win
        for (int row = 0; row < rows; row++)
        {
            int opponent_count = 0;
            int empty_col_idx = -1;
            for (int col = 0; col < columns; col++)
            {
                if (board[row * columns + col] == opponent)
                {
                    opponent_count++;
                }
                else if (board[row * columns + col] == 0)
                {
                    empty_col_idx = col;
                }
            }
            if (opponent_count == columns - 1 && empty_col_idx != -1)
            {
                board[row * columns + empty_col_idx] = player;
                return;
            }
        }

        // Check for a potential vertical win
        for (int col = 0; col < columns; col++)
        {
            int opponent_count = 0;
            int empty_row_idx = -1;
            for (int row = 0; row < rows; row++)
            {
                if (board[row * columns + col] == opponent)
                {
                    opponent_count++;
                }
                else if (board[row * columns + col] == 0)
                {
                    empty_row_idx = row;
                }
            }
            if (opponent_count == rows - 1 && empty_row_idx != -1)
            {
                board[empty_row_idx * columns + col] = player;
                return;
            }
        }

        // No block needed, make a move
        for (int col = 0; col < columns; col++)
        {
            for (int i = rows - 1; i >= 0; i--)
            {
                if (board[col * rows + i] == 0)
                {
                    board[col * rows + i] = player;
                    return;
                }
            }
        }
    }
}

void printBoard(const int *board, int rows, int columns)
{
    printf("Board:\n");
    for (int i = 0; i < rows; ++i)
    {
        for (int j = 0; j < columns; ++j)
        {
            if (board[i * columns + j] == 1){
                printf("%s ", "O");
            }
            else if (board[i * columns + j] == 2){
                printf("%s ", "X");
            }
            else{
                printf("%s ", "-");
            }
        }
        printf("\n");
    }
    printf("\n");
}

bool checkWin(const int *board, int player, int rows, int columns)
{
    int winCondition = rows;

    for (int row = 0; row < rows; row++)
    {
        for (int col = 0; col <= columns - winCondition; col++)
        {
            bool win = true;
            for (int k = 0; k < winCondition; k++)
            {
                if (board[row * columns + col + k] != player)
                {
                    win = false;
                    break;
                }
            }
            if (win)
            {
                printf("Player %d wins horizontally at row %d, column %d\n", player, row, col);
                return true;
            }
        }
    }

    for (int col = 0; col < columns; col++)
    {
        for (int row = 0; row <= rows - winCondition; row++)
        {
            bool win = true;
            for (int k = 0; k < winCondition; k++)
            {
                if (board[(row + k) * columns + col] != player)
                {
                    win = false;
                    break;
                }
            }
            if (win)
            {
                printf("Player %d wins vertically at row %d, column %d\n", player, row, col);
                return true;
            }
        }
    }

    for (int row = 0; row <= rows - winCondition; row++)
    {
        for (int col = 0; col <= columns - winCondition; col++)
        {
            bool win = true;
            for (int k = 0; k < winCondition; k++)
            {
                if (board[(row + k) * columns + col + k] != player)
                {
                    win = false;
                    break;
                }
            }
            if (win)
            {
                printf("Player %d wins diagonally (\\) at row %d, column %d\n", player, row, col);
                return true;
            }
        }
    }

    for (int row = 0; row <= rows - winCondition; row++)
    {
        for (int col = winCondition - 1; col < columns; col++)
        {
            bool win = true;
            for (int k = 0; k < winCondition; k++)
            {
                if (board[(row + k) * columns + col - k] != player)
                {
                    win = false;
                    break;
                }
            }
            if (win)
            {
                printf("Player %d wins diagonally (/) at row %d, column %d\n", player, row, col);
                return true;
            }
        }
    }

    return false;
}

int main(int argc, char *argv[])
{
    int rows = DEFAULT_ROWS;
    int columns = DEFAULT_COLUMNS;

    if (argc >= 2)
    {
        rows = atoi(argv[1]);
        columns = atoi(argv[1]);
    }

    printf("Board size: %dx%d\n", rows, columns);

    int nDevices;
    hipGetDeviceCount(&nDevices);

    int device_1 = 0;
    int device_2 = (nDevices >= 2) ? 1 : 0;

    if (nDevices < 2)
    {
        printf("Warning: Only one GPU available.\n");
    }

    int *board_gpu1, *board_gpu2;
    hipSetDevice(device_1);
    hipMalloc(&board_gpu1, rows * columns * sizeof(int));
    hipMemset(board_gpu1, 0, rows * columns * sizeof(int));

    hipSetDevice(device_2);
    hipMalloc(&board_gpu2, rows * columns * sizeof(int));
    hipMemset(board_gpu2, 0, rows * columns * sizeof(int));

    int *host_board = (int *)malloc(rows * columns * sizeof(int));
    hipError_t err;

    int maxRounds = rows * columns;
    int round = 0;
    int currentPlayer = 1;

    while (true)
    {
        if (currentPlayer == 1)
        {
            unsigned long long seed = time(NULL) + round;
            randomMove<<<1, 1>>>(board_gpu1, currentPlayer, rows, columns, seed);
            err = hipDeviceSynchronize();
            if (err != hipSuccess)
            {
                printf("CUDA error in randomMove: %s\n", hipGetErrorString(err));
                return -1;
            }

            hipMemcpy(host_board, board_gpu1, rows * columns * sizeof(int), hipMemcpyDeviceToHost);
            printBoard(host_board, rows, columns);
            if (checkWin(host_board, currentPlayer, rows, columns))
            {
                printf("Player %d wins!\n", currentPlayer);
                break;
            }

            hipMemcpy(board_gpu2, board_gpu1, rows * columns * sizeof(int), hipMemcpyDeviceToDevice);
            currentPlayer = 2;
        }
        else
        {
            lookaheadMove<<<1, 1>>>(board_gpu2, currentPlayer, rows, columns);
            err = hipDeviceSynchronize();
            if (err != hipSuccess)
            {
                printf("CUDA error in lookaheadMove: %s\n", hipGetErrorString(err));
                break;
            }

            hipMemcpy(host_board, board_gpu2, rows * columns * sizeof(int), hipMemcpyDeviceToHost);
            printBoard(host_board, rows, columns);
            if (checkWin(host_board, currentPlayer, rows, columns))
            {
                printf("Player %d wins!\n", currentPlayer);
                break;
            }

            hipMemcpy(board_gpu1, board_gpu2, rows * columns * sizeof(int), hipMemcpyDeviceToDevice);
            currentPlayer = 1;
        }

        round++;
        if (round >= maxRounds)
        {
            printf("Draw! No moves left.\n");
            break;
        }
    }

    hipFree(board_gpu1);
    hipFree(board_gpu2);
    free(host_board);

    printf("Game over.\n");
    return 0;
}
